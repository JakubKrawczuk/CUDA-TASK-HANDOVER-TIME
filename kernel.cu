#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <chrono>

#define TEST_RUN_COUNT 10
#define IGNORE_FIRST_TESTS 2
#define VERBOSE false

hipError_t addWithCuda(float *c, float*a, float *b, unsigned int size);

__global__ void addKernel(float *c, float *a, float *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

__global__ void addKernelBlock(float* c, float* a, float* b) {
    int ti = threadIdx.x;
    int bi = blockIdx.x;
    int bd = blockDim.x;
    int i = bd * bi + ti;
    c[i] = a[i] + b[i];
}

void generateData(float* arr, int arraySize, float mul);

float minMaxAvg(char op, long long time) {
    static int no = 0;
    static long long min;
    static long long max;
    static long long sum=0;

    if (op == 'r') {
        //reset
        no = 0;
        sum = 0;
    }
    else if (op == 'e') {
        printf("\tmin: %d\n", min);
        printf("\tmax: %d\n", max);
        float avg = (float)sum / (TEST_RUN_COUNT - IGNORE_FIRST_TESTS);
        printf("\tavg: %f\n", avg);
        return avg;
    }
    else {
        if (no == 0) {
            min = time;
            max = time;
        }
        else {
            if (min > time) min = time;
            if (max < time) max = time;
        }

        if (no >= IGNORE_FIRST_TESTS) {
            sum += time;
        }        

        if(VERBOSE) printf("\t\t%d\n", time);
        no++;
    }
    return 0;
}

int main()
{
    const int arraySize = 50000; //test value 100 and 50000
    float a[arraySize] = { 0 };
    float b[arraySize] = { 0 };
    float c[arraySize] = { 0 };

    generateData(a, arraySize, 1);
    generateData(b, arraySize, 0.5);

    /*
    * Time overhead caused by getting time and first variable declaration
    * In my case this value veries from 0 to 300 ns
    */
    auto start = std::chrono::high_resolution_clock::now();
    auto finish = std::chrono::high_resolution_clock::now();
    /*printf("Timer overhead check 1: %d ns (likely 0-300ns)\n",
        std::chrono::duration_cast<std::chrono::nanoseconds>(finish - start).count());

    printf("Timer overhead check 2 ns (likely 0-300ns)\n");
    */

    //time overhead caused by getting time (also varies from 0 to 300 ns)
    if (VERBOSE) {
        for (int i = 0; i < TEST_RUN_COUNT; i++) {
            start = std::chrono::high_resolution_clock::now();
            finish = std::chrono::high_resolution_clock::now();
            minMaxAvg('-', std::chrono::duration_cast<std::chrono::nanoseconds>(finish - start).count());
        }
        minMaxAvg('e', 0);
        minMaxAvg('r', 0);
    }

    /*
    //printf overhead
    start = std::chrono::high_resolution_clock::now();
    printf("Fake value only for test: %d ns\n",
        std::chrono::duration_cast<std::chrono::nanoseconds>(finish - start).count());
    finish = std::chrono::high_resolution_clock::now();
    printf("Printf overhead: %d ns (likely ~32000ns)\n",
        std::chrono::duration_cast<std::chrono::nanoseconds>(finish - start).count());
    */

    /*
    * CUDA
    */
    printf("\n");
    // Add vectors in parallel.

    //start = std::chrono::high_resolution_clock::now();
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }
    //finish = std::chrono::high_resolution_clock::now();

    /*printf("CUDA in %d ns - contains additional overhead from printf and time check look at partial values\n",
       std::chrono::duration_cast<std::chrono::nanoseconds>(finish - start).count());
    */

    /*
    printf("\n");
    for (int i = 0; i < arraySize; i++) {
        printf("%f; ", c[i]);
    }
    printf("\n");
    */

    /*
    * CPU
    */
    printf("CPU in ns\n");
    for (int i = 0; i < TEST_RUN_COUNT; i++) {
        start = std::chrono::high_resolution_clock::now();
        for (int i = 0; i < arraySize; i++) {
            c[i] = a[i] + b[i];
        }
        finish = std::chrono::high_resolution_clock::now();
        minMaxAvg('-', std::chrono::duration_cast<std::chrono::nanoseconds>(finish - start).count());
    }
    minMaxAvg('e', TEST_RUN_COUNT);
    minMaxAvg('r', 0);

    printf("\n\nFOR 100 elements\n");
    //FOR 100
    cudaStatus = addWithCuda(c, a, b, 100);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("CPU in ns\n");
    for (int i = 0; i < TEST_RUN_COUNT; i++) {
        start = std::chrono::high_resolution_clock::now();
        for (int i = 0; i < 100; i++) {
            c[i] = a[i] + b[i];
        }
        finish = std::chrono::high_resolution_clock::now();
        minMaxAvg('-', std::chrono::duration_cast<std::chrono::nanoseconds>(finish - start).count());
    }
    minMaxAvg('e', TEST_RUN_COUNT);
    minMaxAvg('r', 0);
    

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(float*c, float *a, float *b, unsigned int size)
{
    hipError_t cudaStatus;
    float timeSum = 0;

    auto start = std::chrono::high_resolution_clock::now();
    auto finish = std::chrono::high_resolution_clock::now();

    // Choose which GPU to run on, change this on a multi-GPU system.
    printf("CUDA - device choosed ns\n");
    for (int i = 0; i < TEST_RUN_COUNT; i++) {
        start = std::chrono::high_resolution_clock::now();
        cudaStatus = hipSetDevice(0);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
            goto Error;
        }
        finish = std::chrono::high_resolution_clock::now();
        minMaxAvg('-', std::chrono::duration_cast<std::chrono::nanoseconds>(finish - start).count());
    }
    timeSum += minMaxAvg('e', 0);
    minMaxAvg('r', 0);

    

    /*
    * MEMORY
    */
    printf("CUDA - memory prepare in ns) \n");
    for (int i = 0; i < TEST_RUN_COUNT; i++) {
        start = std::chrono::high_resolution_clock::now();
        float* dev_a = 0;
        float* dev_b = 0;
        float* dev_c = 0;

        // Allocate GPU buffers for three vectors (two input, one output)    .
        cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(float));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(float));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(float));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto Error;
        }

        // Copy input vectors from host memory to GPU buffers.
        cudaStatus = hipMemcpy(dev_a, a, size * sizeof(float), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }

        cudaStatus = hipMemcpy(dev_b, b, size * sizeof(float), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }

        finish = std::chrono::high_resolution_clock::now();
        minMaxAvg('-', std::chrono::duration_cast<std::chrono::nanoseconds>(finish - start).count());

        hipFree(dev_c);
        hipFree(dev_a);
        hipFree(dev_b);
    }
    timeSum += minMaxAvg('e', 0);
    minMaxAvg('r', 0);
    
    //for real this time
    float* dev_a = 0;
    float* dev_b = 0;
    float* dev_c = 0;

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    

    /*
    * Claculations
    */
    printf("CUDA - calculations in ns\n");
    for (int i = 0; i < TEST_RUN_COUNT; i++) {
        start = std::chrono::high_resolution_clock::now();

        if (size <= 256) {
            //one Block
            // Launch a kernel on the GPU with one thread for each element.
            addKernel << <1, size >> > (dev_c, dev_a, dev_b);
        }
        else {
            //multiple blocks
            int blocks_count = size / 256 + ((size % 256 == 0) ? 0 : 1);
            addKernelBlock << <blocks_count, 256 >> > (dev_c, dev_a, dev_b);
        }

        // Check for any errors launching the kernel
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            goto Error;
        }

        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
            goto Error;
        }

        finish = std::chrono::high_resolution_clock::now();
        minMaxAvg('-', std::chrono::duration_cast<std::chrono::nanoseconds>(finish - start).count());
    }
    timeSum += minMaxAvg('e', 0);
    minMaxAvg('r', 0);
       


    /*
    * Value return
    */
    printf("CUDA - return in ns\n");
    for (int i = 0; i < TEST_RUN_COUNT; i++) {
        start = std::chrono::high_resolution_clock::now();

        // Copy output vector from GPU buffer to host memory.
        cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }

        finish = std::chrono::high_resolution_clock::now();
        minMaxAvg('-', std::chrono::duration_cast<std::chrono::nanoseconds>(finish - start).count());
    }
    timeSum += minMaxAvg('e', 0);
    minMaxAvg('r', 0);

    printf("CUDA - all avg: %f ns\n", timeSum);
       

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}

void generateData(float* arr, int arraySize, float mul) {
    for (int i = 0; i < arraySize; i++) {
        arr[i] = (i + 1) * mul;
    }
}